
#include <hip/hip_runtime.h>
#if GOOGLE_CUDA

#include "sum_coherencies_op_gpu.cuh"

MONTBLANC_NAMESPACE_BEGIN
MONTBLANC_SUM_COHERENCIES_NAMESPACE_BEGIN

// Register a GPU kernel for SumCoherencies that handles floats
REGISTER_KERNEL_BUILDER(
    Name("SumCoherencies")
    .TypeConstraint<float>("FT")
    .TypeConstraint<tensorflow::complex64>("CT")
    .Device(tensorflow::DEVICE_GPU),
    SumCoherencies<GPUDevice, float, tensorflow::complex64>);

// Register a GPU kernel for SumCoherencies that handles doubles
REGISTER_KERNEL_BUILDER(
    Name("SumCoherencies")
    .TypeConstraint<double>("FT")
    .TypeConstraint<tensorflow::complex128>("CT")
    .Device(tensorflow::DEVICE_GPU),
    SumCoherencies<GPUDevice, double, tensorflow::complex128>);

MONTBLANC_SUM_COHERENCIES_NAMESPACE_STOP
MONTBLANC_NAMESPACE_STOP

#endif // #if GOOGLE_CUDA
