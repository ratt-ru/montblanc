#include "hip/hip_runtime.h"
/* A file to test importing C modules for handling arrays to Python */

#include "Python.h"
#include "arrayobject.h"
#include <math.h>
#include <assert.h>
#include <stdio.h>

#include "segreduce.h"

#include <hip/hip_runtime.h>
#include <moderngpu.cuh>

// Need a plus operator for this
inline __host__ __device__ double2 operator+(const double2 & lhs, const double2 & rhs)
	{ return make_double2(lhs.x + rhs.x, rhs.y + lhs.y); }

inline __host__ __device__ double2 & operator+=(double2 & lhs, const double2 & rhs)
	{ lhs.x += rhs.x; lhs.y += rhs.y; return lhs; }


__global__ void dumb_kernel(int * data, int N)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;

	if(i >= N)
		return;

	data[i] *= 2;
}

template <typename InputIt, typename CsrIt, typename OutputIt, typename T, typename Op>
void seg_reduce_csr_expand(InputIt data_global, CsrIt csr_global, int count,
	int numSegments, OutputIt dest_global, T identity, Op op, hipStream_t stream)
{
	// SegReduceHost (segreducecsr.cuh) STARTS
	typedef typename mgpu::SegReduceNormalTuning<sizeof(T)>::Tuning Tuning;

	// SegReduceInner (segreducecsr.cuh) STARTS
	// TODO: pass the PTX in from PyCUDA somehow
	int2 launch = Tuning::GetLaunchParams(300);
	int NV = launch.x * launch.y;
	int numBlocks = MGPU_DIV_UP(count, NV);
	const int * sources_global = (const int *) 0;

	// PartitionCsrSegReduce (segreduce.cuh) starts here
	int * limitsDevice;

	{
		int numPartitions = numBlocks + 1;
		int numRows = numSegments;
		const int * numRows2 = (const int *) 0;
		hipMalloc(&limitsDevice, sizeof(int)*numPartitions);
		const int NT = 64;

		int numBlocks2 = MGPU_DIV_UP(numPartitions, NT);

		mgpu::KernelPartitionCsrSegReduce<NT><<<numBlocks2, NT, 0, stream>>>(
			count, NV, csr_global, numRows, numRows2, numPartitions,
			limitsDevice);
		// TODO: Add kernel error checking here
	}
	// PartitionCsrSegReduce (segreduce.cuh) ends here

	T * carryOutDevice;
	hipMalloc(&carryOutDevice, sizeof(T)*numBlocks);

	mgpu::KernelSegReduceCsr<Tuning, false>
		<<<numBlocks, launch.x, 0, stream>>>(csr_global,
		sources_global, count, limitsDevice,
		data_global, identity, op, 
		dest_global, carryOutDevice);
	// TODO: Add kernel error checking here

	// SegReduceSpine (segreduce.cuh) starts here
	{
		const int NT = 128;
		int count = numBlocks;
		// redefine numBlocks, but RAII saves us.
		int numBlocks = MGPU_DIV_UP(count, NT);
		int * limits_global = limitsDevice;
		T * carryIn_global = carryOutDevice;
		// redefine carryOutDevice, but RAII saves us.
		T * carryOutDevice;
		hipMalloc(&carryOutDevice, sizeof(T)*numBlocks);

		// Fix-up the segment outputs between the original tiles.
		mgpu::KernelSegReduceSpine1<NT><<<numBlocks, NT, 0, stream>>>(
			limits_global, count, dest_global, carryIn_global, identity, op,
			carryOutDevice);
		// TODO: Add kernel error checking here

		// Loop over the segments that span the tiles of 
		// KernelSegReduceSpine1 and fix those.
		if(numBlocks > 1) {
			mgpu::KernelSegReduceSpine2<NT><<<1, NT, 0, stream>>>(
				limits_global, numBlocks, count, NT, dest_global,
				carryOutDevice, identity, op);
		// TODO: Add kernel error checking here
		}

		hipFree(carryOutDevice);
	}
	// SegReduceSpine (segreduce.cuh) ends here

	hipFree(carryOutDevice);
	hipFree(limitsDevice);

	// SegReduceInner (segreducecsr.cuh) ENDS
} 

template <typename T, typename Op>
PyObject * extract_and_segment(PyObject * self, PyObject * args, PyObject * kw,
	T identity, Op op)
{
	PyObject * value_array;		// pycuda.gpuarray
	PyObject * segment_starts;	// pycuda.gpuarray
	PyObject * segment_sums;	// pycuda.gpuarray
	int device_id;				// int
	PyObject * stream_obj;		// pycuda.driver.Stream

    static char * kwlist[] = {
    	(char *) "value_array",
    	(char *) "segment_starts",
    	(char *) "segment_sums",
    	(char *) "device_id",
    	(char *) "stream",
    	NULL };

    if (!PyArg_ParseTupleAndKeywords(args, kw, "OOOiO", kwlist,
		&value_array,
		&segment_starts,
		&segment_sums,
		&device_id,
		&stream_obj)) return NULL;

	PyObject * value_gpu = PyObject_GetAttrString(value_array, "gpudata");
	PyObject * value_size =  PyObject_GetAttrString(value_array, "size");
	PyObject * segments_gpu = PyObject_GetAttrString(segment_starts, "gpudata");
	PyObject * segments_size =  PyObject_GetAttrString(segment_starts, "size");
	PyObject * segment_sums_gpu = PyObject_GetAttrString(segment_sums, "gpudata");
	PyObject * stream_handle = PyObject_GetAttrString(stream_obj, "handle"); 

	// Extract cuda device pointers, array sizes and stream_id
	// from the Python Objects
	T * value_ptr = (T *) PyInt_AsUnsignedLongLongMask(value_gpu);
	int * segment_ptr = (int *) PyInt_AsUnsignedLongLongMask(segments_gpu);
	T * segment_sums_ptr = (T *) PyInt_AsUnsignedLongLongMask(segment_sums_gpu);
	int n_values =  PyInt_AsLong(value_size);
	int n_segments =  PyInt_AsLong(segments_size);
	hipStream_t stream = (hipStream_t) PyInt_AsUnsignedLongLongMask(stream_handle);

	printf("values address=%p size=%ld\n", value_ptr, n_values);
	printf("segments address=%p size=%ld\n", segment_ptr, n_segments);
	printf("segment sums address=%p\n", segment_sums_ptr);
	printf("device_id=%ld stream=%ld\n", device_id, stream);

	seg_reduce_csr_expand(value_ptr, segment_ptr, n_values,
		n_segments, segment_sums_ptr, identity, op, stream);

	return value_array;
}

#ifdef __cplusplus
extern "C" {
#endif 

/* #### Globals #################################### */
/* ==== Create 1D Carray from PyArray ======================
    Assumes PyArray is contiguous in memory.             */

/* ==== Set up the methods table ====================== */
static PyMethodDef segreduce_Methods[] = {
	{"segmented_reduce_complex128_sum", (PyCFunction) 		segmented_reduce_complex128_sum, METH_VARARGS | METH_KEYWORDS},
	{"segmented_reduce_float32_sum", (PyCFunction) 		segmented_reduce_float32_sum, METH_VARARGS | METH_KEYWORDS},
	{NULL, NULL}     /* Sentinel - marks the end of this structure */
};

/* ==== Initialize the C_test functions ====================== */
// Module name must be _C_arraytest in compile and linked 
void initsegreduce()  {
	(void) Py_InitModule("segreduce", segreduce_Methods);
	import_array();  // Must be present for NumPy.  Called first after above line.
}

static PyObject * segmented_reduce_complex128_sum(PyObject * self, PyObject * args,PyObject * kw)
{
	return extract_and_segment(self, args, kw,
		make_double2(0.,0.), mgpu::plus<double2>());	
}

static PyObject * segmented_reduce_float32_sum(PyObject * self, PyObject * args,PyObject * kw)
{
	return extract_and_segment(self, args, kw,
		0.0f, mgpu::plus<float>());
}

#ifdef __cplusplus
} // extern "C" {
#endif 