#include "hip/hip_runtime.h"
/* A file to test importing C modules for handling arrays to Python */

#include "Python.h"
#include "arrayobject.h"
#include <math.h>
#include <assert.h>
#include <stdio.h>

#include "segreduce.h"

#include <moderngpu.cuh>

// Need a plus operator for this
inline __host__ __device__ double2 operator+(const double2 & lhs, const double2 & rhs)
	{ return make_double2(lhs.x + rhs.x, rhs.y + lhs.y); }

inline __host__ __device__ double2 & operator+=(double2 & lhs, const double2 & rhs)
	{ lhs.x += rhs.x; lhs.y += rhs.y; return lhs; }


#ifdef __cplusplus
extern "C" {
#endif 

/* #### Globals #################################### */
/* ==== Create 1D Carray from PyArray ======================
    Assumes PyArray is contiguous in memory.             */

/* ==== Set up the methods table ====================== */
static PyMethodDef segreduce_Methods[] = {
	{"segmented_reduce_complex128_sum", (PyCFunction) segmented_reduce_complex128_sum, METH_VARARGS | METH_KEYWORDS},
	{NULL, NULL}     /* Sentinel - marks the end of this structure */
};

/* ==== Initialize the C_test functions ====================== */
// Module name must be _C_arraytest in compile and linked 
void initsegreduce()  {
	(void) Py_InitModule("segreduce", segreduce_Methods);
	import_array();  // Must be present for NumPy.  Called first after above line.
}

PyObject * segmented_reduce_complex128_sum(PyObject * self, PyObject * args, PyObject * kw)
{
	PyObject * value_array;		// pycuda.gpuarray
	PyObject * segment_starts;	// pycuda.gpuarray
	PyObject * segment_sums;	// pycuda.gpuarray
	int device_id;				// int
	PyObject * stream;			// pycuda.driver.Stream

    static char *kwlist[] = {
    	"value_array",
    	"segment_starts",
    	"segment_sums",
    	"device_id",
    	"stream",
    	NULL };

    if (!PyArg_ParseTupleAndKeywords(args, kw, "OOO|iO", kwlist,
		&value_array,
		&segment_starts,
		&segment_sums,
		&device_id,
		&stream)) return NULL;

	PyObject * value_gpu = PyObject_GetAttrString(value_array, "gpudata");
	PyObject * value_size =  PyObject_GetAttrString(value_array, "size");
	PyObject * segments_gpu = PyObject_GetAttrString(segment_starts, "gpudata");
	PyObject * segments_size =  PyObject_GetAttrString(segment_starts, "size");
	PyObject * segment_sums_gpu = PyObject_GetAttrString(segment_sums, "gpudata");
	PyObject * stream_handle = PyObject_GetAttrString(stream, "handle"); 

	// Extract cuda device pointers, array sizes and stream_id
	// from the Python Objects
	hipDeviceptr_t value_ptr = (hipDeviceptr_t) value_gpu;
	hipDeviceptr_t segment_ptr = (hipDeviceptr_t) segments_gpu;
	hipDeviceptr_t segment_sums_ptr = (hipDeviceptr_t) segment_sums_gpu;
	long n_values =  PyInt_AsLong(value_size);
	long n_segments =  PyInt_AsLong(segments_size);
	hipStream_t stream_id = (hipStream_t) PyInt_AsLong(stream_handle);

	printf("values address=%p size=%ld\n", value_ptr, n_values);

	printf("segments address=%p size=%ld\n", segment_ptr, n_segments);

	printf("segment sums address=%p\n", segment_sums_ptr);

	printf("device_id=%ld stream=%ld\n", device_id, stream_id);

	// We've finished using all these Python Objects.
	Py_DECREF(value_gpu);
	Py_DECREF(value_size);
	Py_DECREF(segments_gpu);
	Py_DECREF(segments_size);
	Py_DECREF(segment_sums_gpu);
	Py_DECREF(stream_handle);

	mgpu::ContextPtr context_ptr = mgpu::CreateCudaDeviceAttachStream(
		device_id, stream_id);

	mgpu::SegReduceCsr(
		(double2 *) value_ptr,
		(int *) segment_ptr,
		n_values,
		n_segments,
		false,
		(double2 *) segment_sums_ptr,
		make_double2(0.,0.),
		mgpu::plus<double2>(),
		*context_ptr);

	return value_array;
}

#ifdef __cplusplus
} // extern "C" {
#endif 